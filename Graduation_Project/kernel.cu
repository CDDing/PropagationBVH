#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>
#include <hittable_list.h>
#include <ray.h>
#include <sphere.h>
#include <hittable_list.h>
#include <material.h>
#include <vec3.h>
#include <camera.h>
#include <bvh.h>
#include <triangle.h>
#include <obj.h>
#include <assimp/Importer.hpp>
#include <assimp/scene.h>
#include <assimp/postprocess.h>
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>


unsigned char* dbackground_image;
hittable_list** world;
bvh_node** bvh_list;
bool prop = false;
camera** cam;
int object_counts = 1000;
hiprandState* random_state;
int** test;
// convert floating point rgb color to 8-bit integer
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }
__device__ int rgbToInt(float r, float g, float b) {
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}
__device__ int vectorgb(vec3 color) {
	return rgbToInt(color.x() * 255, color.y() * 255, color.z() * 255);
}
__global__ void movCam(camera** ca, int direction, int weight) {
	(*ca)->moveorigin(direction, weight);
}
__global__ void RotateCam(camera** ca, vec3 direction) {
	(*ca)->rotate(direction);
}
__global__ void ManipulateVFOV(camera** ca, int x) {
	(*ca)->changevfov(x);
}
extern "C" void moveCamera(int direction, int weight) {
	movCam << <1, 1 >> > (cam, direction, weight);
}
extern "C" void RotateCamera(int x, int y) {
	RotateCam << <1, 1 >> > (cam, vec3(x, y, 0));
}
extern "C" void manivfov(int x) {
	ManipulateVFOV << <1, 1 >> > (cam, x);
}

__global__ void AddSphere(hittable_list** world,int cnt) {
	hittable** list = (*world)->get_list();
	bvh_node* bvh = (bvh_node*)list[0];
	hiprandState local_rand_state;
	hiprand_init(cnt, 0, 0, &local_rand_state);
	printf("추가");
	bvh->add(new sphere(vec3(RND*10, RND*10, RND*10), 4, new lambertian(vec3(RND, RND, RND)),0));
}

int ballcnt = 0;
extern "C" void GenerateSphere() {
	AddSphere << <1, 1 >> > (world,ballcnt++);
}

__global__ void CalculatePerPixel(hittable_list** world, camera** camera, hiprandState* global_rand_state, unsigned int* g_odata, int imgh, int imgw) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int i = blockIdx.x * bw + tx;
	int j = blockIdx.y * bh + ty;
	int index = i + j * imgh;

	hiprandState local_rand_state = global_rand_state[index];
	vec3 color(0, 0, 0);

	int depth = (*camera)->max_depth;
	int spp = (*camera)->samples_per_pixel;
	float rate = 1 / float(spp);
	ray r = (*camera)->get_ray(&local_rand_state, i, j);
	for (int i = 0; i < spp; i++) {
		color += (*camera)->ray_color(&local_rand_state, r, depth, world);
	}
	color *= rate;
	global_rand_state[index] = local_rand_state;
	g_odata[i + j * imgw] = vectorgb(color);
}
__global__ void initCamera(camera** ca,unsigned char* background_image,int iw,int ih) {
	*ca = new camera(16.0 / 9.0, //종횡비
		1600,                    //이미지 가로길이
		5,                       //픽셀당 샘플수
		5,                      //반사 횟수
		90,                      //시야각
		vec3(-20, 0, 0),         //카메라 위치 
		vec3(0, 0, -1),          //바라보는곳
		vec3(0, 1, 0),           //업벡터
		vec3(0.5f, 0.7f, 1));      //배경색
	(*ca)->Setbackground(background_image,iw,ih);
}
__global__ void initWorld(hittable_list** world, int object_counts) {
	(*world) = new hittable_list(object_counts);
}
#define RND (hiprand_uniform(&local_rand_state))
__global__ void addObjects(hiprandState* global_state, hittable_list** world, int object_counts) {
	hiprand_init(0, 0, 0, &global_state[0]);
	hiprandState local_rand_state = *global_state;
	(*world)->add(new sphere(vec3(0, -1000.0, 0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)),0));

	//(*world)->add(new triangle(vec3(50, 50, 50), vec3(-50, 50, 50), vec3(50, -50, 50), new metal(vec3(0.5, 0.7, 0.8),0)));

	(*world)->add(new sphere(vec3(0, 200, 0), 100, new light(vec3(1, 1, 1)),0));
	int sphere_count = 10;




	for (int a = -sphere_count; a < sphere_count; a++) {
		for (int b = -sphere_count; b < sphere_count; b++) {
			
			float choose_mat = RND;
			vec3 center(a + RND, 0.2, b + RND);
			if (choose_mat < 0.8f) {
				(*world)->add(new sphere(center, 0.2, new lambertian(vec3(RND * RND, RND * RND, RND * RND)),1));
			}
			else if (choose_mat <= 1.0f) {
				(*world)->add(new sphere(center, 0.2, new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.0f/*0.5f * RND*/),1));
			}
			else {
				(*world)->add(new sphere(center, 0.2, new dielectric(1.5),1));
			}
		}
	}
}
__global__ void makeBVH(hiprandState* global_state, hittable_list** world, bvh_node** bvh_list, int object_counts) {
	printf("%d개\n", (*world)->now_size);
	hiprand_init(0, 0, 0, &global_state[0]);
	hiprandState local_rand_state = *global_state;
	(*world) = new hittable_list((hittable*)new bvh_node(world, bvh_list, &local_rand_state), object_counts);

}
__global__ void addTriangle(hittable_list** world, vec3 a, vec3 b, vec3 c, vec3 color) {
	(*world)->add(new triangle(a, b, c, new dielectric(2.0f)));

}
__global__ void Random_Init(hiprandState* global_state, int ih) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;
	unsigned int pixel_index = x + y * ih;
	hiprand_init(pixel_index, 0, 0, &global_state[pixel_index]);
}

__global__ void initMesh(hittable_list** tmp, int obj_counts) {
	(*tmp) = new hittable_list(obj_counts);
}
__global__ void mergeMesh(hittable_list** world, hittable_list** tmp, bvh_node** node, hiprandState* rand_state) {
	hiprand_init(0, 0, 0, &rand_state[0]);
	hiprandState local_rand_state = *rand_state;

	(*world)->add(new bvh_node(tmp, node, &local_rand_state));
}

void ReadOBJ(const char* objlist[], int obj_counts, const vec3 translist[], const vec3 scalelist[]) {
	Assimp::Importer importer;
	for (int c = 0; c < obj_counts; c++) {
		char str[100] = "resource/";
		strcat(str, objlist[c]);
		printf("%s\n", str);
		const aiScene* scene = importer.ReadFile(str, aiProcess_Triangulate | aiProcess_GenSmoothNormals | aiProcess_FlipUVs | aiProcess_CalcTangentSpace);
		if (!scene || scene->mFlags & AI_SCENE_FLAGS_INCOMPLETE || !scene->mRootNode) // if is Not Zero
		{
			printf("Read File Exception\n");
		}
		vec3 translate = translist[c];
		vec3 scale = scalelist[c];
		int cnt = 0;
		for (int i = 0; i < scene->mNumMeshes; i++) {
			auto mesh = scene->mMeshes[i];
			hiprandState* mesh_state;
			hipMalloc(&mesh_state, sizeof(hiprandState));

			bvh_node** node;
			hittable_list** tmp;
			hipMalloc(&tmp, sizeof(hittable_list*));
			int startIdx = 1 << 30;
			while (true) {
				if ((startIdx >> 1) > mesh->mNumFaces) { startIdx >>= 1; }
				else { break; }
			}

			hipMalloc((void**)&node, startIdx * sizeof(bvh_node*));

			initMesh << <1, 1 >> > (tmp, startIdx);

			for (int j = 0; j < mesh->mNumFaces; j++) {
				auto Face = mesh->mFaces[j];
				vec3 a(mesh->mVertices[Face.mIndices[0]].x, mesh->mVertices[Face.mIndices[0]].y, mesh->mVertices[Face.mIndices[0]].z);
				vec3 b(mesh->mVertices[Face.mIndices[1]].x, mesh->mVertices[Face.mIndices[1]].y, mesh->mVertices[Face.mIndices[1]].z);
				vec3 c(mesh->mVertices[Face.mIndices[2]].x, mesh->mVertices[Face.mIndices[2]].y, mesh->mVertices[Face.mIndices[2]].z);
				a *= scale;				b *= scale;				c *= scale;
				a += translate;		b += translate;		c += translate;
				aiMaterial* material = scene->mMaterials[mesh->mMaterialIndex];
				aiColor4D diffuse, specular, ambient;
				aiGetMaterialColor(material, AI_MATKEY_COLOR_AMBIENT, &ambient);
				aiGetMaterialColor(material, AI_MATKEY_COLOR_SPECULAR, &specular);
				aiGetMaterialColor(material, AI_MATKEY_COLOR_DIFFUSE, &diffuse);
				aiColor4D sum = diffuse + specular + ambient;
				vec3 color(sum.r, sum.g, sum.b);
				//color = vec3(1.0f, 0.0f, 0.0f);
				addTriangle << <1, 1 >> > (tmp, a, b, c, color);

			}
			hipDeviceSynchronize();
			mergeMesh << <1, 1 >> > (world, tmp, node, mesh_state);
		}

	}
}

extern "C" void initCuda(dim3 grid, dim3 block, int image_height, int image_width, int pixels) {
	//hipDeviceSetLimit(hipLimitStackSize, 256 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024);
	hipMalloc(&random_state, pixels * sizeof(hiprandState));
	Random_Init << <grid, block, 0 >> > (random_state, image_height);

	const int bytes_per_pixel = 3;
	//배경 이미지 읽기
	auto n = bytes_per_pixel;
	int iw, ih;
	auto background_image = stbi_load("resource/background.png", &iw, &ih, &n, bytes_per_pixel);
	if (background_image == nullptr) {
		printf("이미지 로딩 에러\n");
	}
	hipMalloc(&dbackground_image, iw * ih * bytes_per_pixel);
	hipMemcpy(dbackground_image, background_image, iw * ih * bytes_per_pixel, hipMemcpyHostToDevice);

	//랜덤 초기화
	hipMalloc((void**)&world, sizeof(hittable*));
	initWorld << <1, 1 >> > (world, object_counts); hipDeviceSynchronize();

	//월드 초기화 OBJ 읽기 및 카메라 등
	const char* objlist[] = { "buff-doge.obj" };      //읽을 OBJ 리스트, 및의 배열들과 순서 맞춰야함
	const vec3 translist[] = {
										vec3(10.0f,10.0f,0.0f) };  //위에서 읽을 OBJ를 옮겨주는 벡터
	const vec3 scalelist[] = {
										vec3(5.0f,5.0f,5.0f) };   //위에서 읽을 OBJ의 크기를 바꿔주는 벡터
	//ReadOBJ(objlist, 1, translist, scalelist);

	//여기까지 OBJ 읽기
	hiprandState* objectinit;
	hipMalloc(&objectinit, sizeof(hiprandState));
	addObjects << <1, 1 >> > (objectinit, world, object_counts);
	hipMalloc(&cam, sizeof(camera*));
	initCamera << <1, 1 >> > (cam,dbackground_image,iw,ih);

	hipDeviceSynchronize();        //쿠다커널이 종료될때까지 기다리는 함수. 위의 world에 오브젝트 다 담길때까지 기다림.
	//BVH 생성 중 오브젝트 담기는 것 방지용

	hiprandState* bvh_state;
	hipMalloc(&bvh_state, sizeof(hiprandState));
	hipMalloc((void**)&bvh_list, object_counts * sizeof(bvh_node*));
	makeBVH << <1, 1 >> > (bvh_state, world, bvh_list, object_counts);

}
__global__ void printTraversalCount(hittable_list** world) {
	auto bvh = (bvh_node*)(*world)->get_list()[0];
	printf("평균 순회 횟수 : %f\n",bvh->getTraversal());
}
extern "C" void generatePixel(dim3 grid, dim3 block, int sbytes,
	unsigned int* g_odata, int imgh, int imgw) {
	//Propagation<<<1,1>>>(world,random_state,prop);
	CalculatePerPixel << <grid, block, sbytes >> > (world, cam, random_state, g_odata, imgh, imgw);
	//printTraversalCount << <1, 1 >> > (world);
}